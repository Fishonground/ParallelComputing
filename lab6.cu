
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
//#include <omp.h>


__global__ void map_m1(double* M1_v, int size) {
    //линейный индекс потока
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    //сколько один поток выполняет
    unsigned int threadsNum = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += threadsNum) {
        M1_v[i] = tanh(M1_v[i]);
    }
}

__global__ void map_m2(double* M2_v, double* M3_v, int size) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int threadsNum = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += threadsNum) {
        if (i == 0) {
            M2_v[i] = log(fabs(tan(M2_v[i])));
        } else {
            M2_v[i] = log(fabs(tan(M2_v[i] + M3_v[i - 1])));
        }
    }
}


__global__ void merge(const double* M1_v, double* M2_v, int size) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int threadsNum = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += threadsNum) {
        M2_v[i] = (double) M1_v[i] * M2_v[i];
    }
}






void swap(double *x, double *y){
    double t = *x;
    *x = *y;
    *y = t;
}
void heapify(double arr[], int n, int i)
{
    int largest = i;
    int l = 2*i + 1; // левый = 2*i + 1
    int r = 2*i + 2; // правый = 2*i + 2
    if (l < n && arr[l] > arr[largest])
        largest = l;
    if (r < n && arr[r] > arr[largest])
        largest = r;
    if (largest != i)
    {
        swap(&arr[i], &arr[largest]);
        heapify(arr, n, largest);
    }
}

void heapSort(double arr[], int n)
{
    for (int i = n / 2 - 1; i >= 0; i--)
        heapify(arr, n, i);
        
    for (int i=n-1; i>=0; i--)
    {
        swap(&arr[0], &arr[i]);
        heapify(arr, i, 0);
    }
}

int main(int argc, char* argv[])
{
    int j, N;
    struct timeval T1, T2;
    long delta_ms;
    N = atoi(argv[1]); /* N равен первому параметру командной строки */
    
    
    //omp_set_num_threads(amount);
    gettimeofday(&T1, NULL); /* запомнить текущее время T1 */
    double *M1;  // указатель на массив
    M1 = (double*)malloc(N * sizeof(double));
    double *M2;  // указатель на массив
    M2 = (double*)malloc(N/2 * sizeof(double));
    double *M3;  // указатель на массив
    M3 = (double*)malloc(N/2 * sizeof(double));
    
    //lab 6
    
    
    double *M1_v, *M2_v, *M3_v;
    //выделение памяти на устройстве
    hipMalloc(&M1_v, sizeof(double) * N);
    hipMalloc(&M2_v, sizeof(double) * N / 2);
    hipMalloc(&M3_v, sizeof(double) * N / 2);

    gettimeofday(&T1, nullptr); //запомнить текущее время T1

    //для сихронизации потоков
    hipEvent_t syncEvent;
    hipEventCreate(&syncEvent);    //Создаем event
    hipEventRecord(syncEvent, nullptr);  //Записываем event

    //расчёт gridSize и blockSize для m1
    int gridSize, blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, map_m1, 0, N);
    gridSize = (N + blockSize - 1) / blockSize;

    //расчёт gridSize и blockSize для m2
    int minGridSize2, blockSize2, gridSize2;
    hipOccupancyMaxPotentialBlockSize(&minGridSize2, &blockSize2, map_m2, 0, N / 2);
    gridSize2 = (N / 2  + blockSize2 - 1) / blockSize2;
    
    
	
    for (j=0; j<100; j++)  /*100 экспериментов */
    {
        int i = 0;
		unsigned int seed = j;
		
		double sum = 0;
		double del = 0;
		
		//#pragma omp parallel default(none) private(i) shared(M1,M2,M3,N,seed,sum,del)
		//{
       
       
       		//GENERATE
       		
       		
			//#pragma omp single
			for (i = 0; i<N; i++)
			{
				M1[i] = rand_r(&seed)%420;
			}
			
			//#pragma omp single
			for (i = 0; i<N/2; i++)
			{
				M3[i] = M2[i] = 420+rand_r(&seed)%3780;
			}
			
			//копирование данных после инициализации
        		hipMemcpy(M1_v, M1, sizeof(double) * N, hipMemcpyHostToDevice);
        		hipMemcpy(M2_v, M2, sizeof(double) * N / 2, hipMemcpyHostToDevice);
        		hipMemcpy(M3_v, M3, sizeof(double) * N / 2, hipMemcpyHostToDevice);
			
		
			//MAP
			
			
			map_m1<<<gridSize, blockSize>>>(M1_v, N);
        		map_m2<<<gridSize2, blockSize2>>>(M2_v, M3_v, N / 2);
        		
        		hipEventSynchronize(syncEvent);  //Синхронизируем event

        		hipMemcpy(M1, M1_v, sizeof(double) * N, hipMemcpyDeviceToHost);
        		hipMemcpy(M2, M2_v, sizeof(double) * N / 2, hipMemcpyDeviceToHost);
			
			//#pragma omp for 
			//for (i = 0; i<N; i++)
			//{
			//	M1[i] = tanh(M1[i]);
			//}
			
			//#pragma omp single
			M2[0] = log(fabs(tan(M2[0])));
			
			//#pragma omp for
			//for (i = 1; i<N/2; i++)
			//{
			//	M2[i] += M3[i-1];
			//	M2[i] = log(fabs(tan(M2[i])));
			//}
			
			//MERGE
			
			merge<<<gridSize2, blockSize2>>>(M1_v, M2_v, N / 2);

			hipEventSynchronize(syncEvent);  //Синхронизируем event
			hipMemcpy(M2, M2_v, sizeof(double) * N / 2, hipMemcpyDeviceToHost);
			
			//#pragma omp for 
			//for (i = 0; i<N/2; i++)
			//{		
			//	M2[i] = M2[i] * M1[i];
			//}
			
			
			
			//SORT
			
			//Can't be parallelized cause it's an operation of sorting
			//#pragma omp single
			heapSort(M2,N/2);
		
			hipEventSynchronize(syncEvent);  //Синхронизируем event
			
			//REDUCE
			
			
			//Can't be parallelized cause it's an operation of searching FIRST not Null element
			//#pragma omp single
			for (i = 0; i<N/2; i++) {
				if (M2[i]!=0) {
					del = M2[i];
					break;
				}
			}
			
			//#pragma omp for	reduction(+:sum)		
			for (i = 0; i<N/2; i++) {
				if ((int)(M2[i]/del)%2 == 0) {
					double x = sin(M2[i]);
					sum += x;
				}
			}
			
			hipEventSynchronize(syncEvent);  //Синхронизируем event
		}
		
        //printf("\nN=%d. %f", N, sum);
    //}
    
    hipEventDestroy(syncEvent);

    hipFree(M1);
    hipFree(M2);
    hipFree(M3);

    free(M1);
    free(M2);
    free(M3);
    
    
    
    gettimeofday(&T2, NULL); /* запомнить текущее время T2 */
    delta_ms = 1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec) /1000;
    printf("\nN=%d. Milliseconds passed: %ld", N, delta_ms); /* T2 - T1 */
    return 0;
}
