
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>

__global__ void map_m1(double* M1_v, int size) {
    //линейный индекс потока
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    //сколько один поток выполняет
    unsigned int threadsNum = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += threadsNum) {
        M1_v[i] = tanh(M1_v[i]);
    }
}

__global__ void map_m2(double* M2_v, double* M3_v, int size) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int threadsNum = blockDim.x * gridDim.x;
    
    for (unsigned int i = id; i < size; i += threadsNum) {
        if (i == 0) {
            M2_v[i] = log(fabs(tan(M2_v[i])));
        } else {
            M2_v[i] = log(fabs(tan(M2_v[i] + M3_v[i-1])));
        }
    }
}

__global__ void merge(const double* M1_v, double* M2_v, int size) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int threadsNum = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += threadsNum) {
        M2_v[i] = (double) M1_v[i] * M2_v[i];
    }
}


void swap(double *x, double *y){
    double t = *x;
    *x = *y;
    *y = t;
}
void heapify(double arr[], int n, int i)
{
    int largest = i;
    int l = 2*i + 1; // левый = 2*i + 1
    int r = 2*i + 2; // правый = 2*i + 2
    if (l < n && arr[l] > arr[largest])
        largest = l;
    if (r < n && arr[r] > arr[largest])
        largest = r;
    if (largest != i)
    {
        swap(&arr[i], &arr[largest]);
        heapify(arr, n, largest);
    }
}

void heapSort(double arr[], int n)
{
    for (int i = n / 2 - 1; i >= 0; i--)
        heapify(arr, n, i);
        
    for (int i=n-1; i>=0; i--)
    {
        swap(&arr[0], &arr[i]);
        heapify(arr, i, 0);
    }
}

int main(int argc, char* argv[])
{
    int j, N;
    struct timeval T1, T2, T3, T4, T5;
    long delta_ms,delta_ms2,delta_ms3,delta_ms4;
    N = atoi(argv[1]); /* N равен первому параметру командной строки */
    
    //omp_set_num_threads(amount);
    gettimeofday(&T1, NULL); /* запомнить текущее время T1 */
    double *M1;  // указатель на массив
    M1 = (double*)malloc(N * sizeof(double));
    double *M2;  // указатель на массив
    M2 = (double*)malloc(N/2 * sizeof(double));
    double *M3;  // указатель на массив
    M3 = (double*)malloc(N/2 * sizeof(double));
    
    //lab 6
    double *M1_v, *M2_v, *M3_v;
    //выделение памяти на устройстве
    hipMalloc(&M1_v, sizeof(double) * N);
    hipMalloc(&M2_v, sizeof(double) * N / 2);
    hipMalloc(&M3_v, sizeof(double) * N / 2);
    //gettimeofday(&T1, nullptr); //запомнить текущее время T1
    //printf("%ld, %ld\n", T1.tv_sec, T1.tv_usec);
    //printf("T1: %ld, %ld\n", T1.tv_sec, T1.tv_usec);
	
    //для сихронизации потоков
    hipEvent_t syncEvent;
    hipEventCreate(&syncEvent);    //Создаем event
    hipEventRecord(syncEvent, NULL);  //Записываем event
    //расчёт gridSize и blockSize для m1
    int gridSize, blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, map_m1, 0, N);
    gridSize = (N + blockSize - 1) / blockSize;
    //расчёт gridSize и blockSize для m2
    int minGridSize2, blockSize2, gridSize2;
    hipOccupancyMaxPotentialBlockSize(&minGridSize2, &blockSize2, map_m2, 0, N / 2);
    gridSize2 = (N / 2  + blockSize2 - 1) / blockSize2;
    float ttime = 0;
	
    for (j=0; j<100; j++)  /*100 экспериментов */
    {
        int i = 0;
	unsigned int seed = j;
	double sum = 0;
	double del = 0;
       gettimeofday(&T1, nullptr);
       //GENERATE
	for (i = 0; i<N; i++)
	{
		M1[i] = rand_r(&seed)%420;
	}
	
	for (i = 0; i<N/2; i++)
	{
		M3[i] = M2[i] = 420+rand_r(&seed)%3780;
	}
	
	
	gettimeofday(&T2, nullptr);
	
	//копирование данных после инициализации
	hipMemcpy(M1_v, M1, sizeof(double) * N, hipMemcpyHostToDevice);
	hipMemcpy(M2_v, M2, sizeof(double) * N / 2, hipMemcpyHostToDevice);
	hipMemcpy(M3_v, M3, sizeof(double) * N / 2, hipMemcpyHostToDevice);
	
	
	hipEvent_t start, stop;
	
	hipEventCreate (&start);
	hipEventCreate (&stop);
	hipEventRecord (start, 0);
	//MAP
	map_m1<<<gridSize, blockSize>>>(M1_v, N);
	map_m2<<<gridSize2, blockSize2>>>(M2_v, M3_v, N / 2);
	hipEventSynchronize(syncEvent);  //Синхронизируем event
	hipMemcpy(M1, M1_v, sizeof(double) * N, hipMemcpyDeviceToHost);
	hipMemcpy(M2, M2_v, sizeof(double) * N / 2, hipMemcpyDeviceToHost);
	
	
	
	//gettimeofday(&T3, NULL);
	
    	//printf("T4: %ld, %ld\n", T4.tv_sec, T4.tv_usec);
	//MERGE
	merge<<<gridSize2, blockSize2>>>(M1_v, M2_v, N / 2);
	hipEventSynchronize(syncEvent);  //Синхронизируем event
	hipMemcpy(M2, M2_v, sizeof(double) * N / 2, hipMemcpyDeviceToHost);
	
	hipEventRecord(stop,0);
	hipEventSynchronize (stop);
	hipEventElapsedTime (&ttime, start, stop);
	//printf("%.32f\n",ttime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	
	
	gettimeofday(&T3, nullptr);
	//printf("T2: %ld, %ld\n", T2.tv_sec, T2.tv_usec);
	
	//SORT
	heapSort(M2,N/2);
	hipEventSynchronize(syncEvent); 
	
	
	//REDUCE
	for (i = 0; i<N/2; i++) {
		if (M2[i]!=0) {
			del = M2[i];
			break;
		}
	}
	for (i = 0; i<N/2; i++) {
		if ((int)(M2[i]/del)%2 == 0) {
			double x = sin(M2[i]);
			sum += x;
		}
	}
	hipEventSynchronize(syncEvent);  //Синхронизируем event
	gettimeofday(&T4, nullptr);
	
	//printf("N=%d. %f\n", N, sum);
	//printf("test: %d\n", T2.tv_sec-T1.tv_sec);
    	delta_ms2 = 1000*(T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec) /1000;
    	//printf("test: %f\n", delta_ms2);
    	delta_ms3 = 1000*(T3.tv_sec - T2.tv_sec) + (T3.tv_usec - T2.tv_usec) /1000;
    	//delta_ms4 = 1000*(T4.tv_sec - T3.tv_sec) + (T4.tv_usec - T3.tv_usec) /1000;
    	//delta_ms3 = 0;
    	//delta_ms4 = 0.0;
    	//printf("%.16f\n", ttime);
    	printf("\nN=%d. Milliseconds passed: %ld, %ld \n", N, delta_ms2, delta_ms3);
    	//printf("\nN=%d. Milliseconds passed: %ld, %ld, %ld\n", N, delta_ms2, delta_ms3, delta_ms4); /* T2 - T1 */
    }
		
    
    hipEventDestroy(syncEvent);
    hipFree(M1);
    hipFree(M2);
    hipFree(M3);
    free(M1);
    free(M2);
    free(M3);
    
    gettimeofday(&T5, NULL); /* запомнить текущее время T2 */
    //delta_ms = 1000*(T5.tv_sec - T3.tv_sec) + (T5.tv_usec - T3.tv_usec) /1000;
    delta_ms = 1000*(T5.tv_sec - T1.tv_sec) + (T5.tv_usec - T1.tv_usec) /1000;
    //delta_ms2 = 1000*(T4.tv_sec - T3.tv_sec) + (T4.tv_usec - T3.tv_usec) /1000;
    //delta_ms3 = 1000*(T5.tv_sec - T4.tv_sec) + (T5.tv_usec - T4.tv_usec) /1000;
    printf("\nN=%d. Milliseconds passed: %ld \n", N, delta_ms); /* T2 - T1 */
    return 0;
}
